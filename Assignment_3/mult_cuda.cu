#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrix_multiply(float *A, float *B, float *C, int ROWS, int COLS)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ROWS && j < COLS)
    {
        float sum = 0.0f;
        for (int k = 0; k < COLS; ++k)
        {
            sum += A[i * COLS + k] * B[k * COLS + j];
        }
        C[i * COLS + j] = sum;
    }
}

int main()
{
    int ROWS = 2000; // number of rows
    int COLS = 2000; // number of columns 

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *A, *B, *C;       // matrices
    float *d_A, *d_B, *d_C; // device matrices
    size_t sizeA = ROWS * COLS * sizeof(float);
    size_t sizeB = COLS * COLS * sizeof(float);
    size_t sizeC = ROWS * COLS * sizeof(float);

    // allocate memory for host matrices
    A = (float *)malloc(sizeA);
    B = (float *)malloc(sizeB);
    C = (float *)malloc(sizeC);

    // initialize host matrices with random data
    for (int i = 0; i < ROWS; ++i)
    {
        for (int j = 0; j < COLS; ++j)
        {
            A[i * COLS + j] = rand() / (float)RAND_MAX;
        }
    }
    for (int i = 0; i < COLS; ++i)
    {
        for (int j = 0; j < COLS; ++j)
        {
            B[i * COLS + j] = rand() / (float)RAND_MAX;
        }
    }

    // allocate memory for device matrices
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    // copy host matrices to device
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    // set grid and block sizes for kernel launch
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((COLS + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ROWS + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // launch kernel to perform matrix multiplication on device
    hipEventRecord(start);
    matrix_multiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, ROWS, COLS);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time with CUDA: %f seconds\n", milliseconds / 1000.0f);

    // copy result matrix from device to host
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    /*
    // print result matrix
    printf("Result matrix:\n");
    for (int i = 0; i < ROWS; ++i) {
        for (int j = 0; j < K; ++j) {
            printf("%f ", C[i*K+j]);
        }
        printf("\n");
    }*/

    // free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
